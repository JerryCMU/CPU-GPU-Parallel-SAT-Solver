#include <stdio.h>
#include <chrono>
#include <unistd.h>
#include <iostream>

#include <hip/hip_runtime.h>

#include <driver_functions.h>

__device__ bool getValue(uint64_t assignment, int variable) {
    if (variable < 0) {
        int position = -1 * variable - 1;
        return ((assignment >> position) & 0x1) == 0;
    } else {
        int position = variable - 1;
        return ((assignment >> position) & 0x1) == 1;
    }
}

__device__ bool solve(int *CNF, uint64_t assignment, int c_count) {
    for (int i = 0; i < c_count; i++) {
        int c1 = CNF[3*i];
        int c2 = CNF[3*i+1];
        int c3 = CNF[3*i+2];

        if (getValue(assignment, c1) || getValue(assignment, c2) || getValue(assignment,c3)) {
            continue;
        }

        return false;
    }

    return true;
}

__global__ void
solve_kernel(int* CNF, size_t thread_size, int c_count, bool *flag) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    uint64_t start_assignment = index * thread_size;
    uint64_t end_assignment = (index + 1) * thread_size;

    for (uint64_t assignment = start_assignment; assignment < end_assignment; assignment++) {
        if (solve(CNF, assignment, c_count)) {
            *flag = true;
            break;
        }
    }
}

void
SATCuda(int* CNF, int v_count, int c_count) {

    uint64_t base = 1;
    uint64_t problem_size = base << v_count;

    // compute number of blocks and threads per block
    const int threadsCount = 1<<5;
    const int threadsPerBlock = 32;
    const int blocks = (threadsCount) / threadsPerBlock;

    size_t thread_size = problem_size / threadsCount;
    
    bool flag = false;

    int* device_CNF;
    bool* device_flag;

    int vectorBytes = sizeof(int) * 3 * c_count;
    hipMalloc(&device_CNF, vectorBytes);
    hipMalloc(&device_flag, sizeof(bool));

    // start timing after allocation of device memory
    auto start = std::chrono::high_resolution_clock::now();
    hipMemcpy(device_CNF, CNF, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(device_flag, &flag, sizeof(bool), hipMemcpyHostToDevice);
    // start timing after data copy

    // run kernel
    solve_kernel<<<blocks, threadsPerBlock>>>(device_CNF, thread_size, c_count, device_flag);
    hipDeviceSynchronize();

    // end timining after all threads are computed
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop-start);
    std::cout << "CUDA execution time " << duration.count() << " ms\n"; 

    hipMemcpy(&flag, device_flag, sizeof(bool), hipMemcpyDeviceToHost);

    if (flag) {
        std::cout << "SOLVABLE!\n";
    } else {
        std::cout << "UNSOLVABLE!\n";
    }

    // end timing after result has been copied back into host memory

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    // TODO free memory buffers on the GPU
    hipFree(device_CNF);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
